
#include "hip/hip_runtime.h"


#include <stdio.h>



__global__ void add3(float *val1, float *val2, int *num_elem)
{
	int i = threadIdx.x;
	val1[i] += val2[i];
}

__global__ void sub3(float *val1, float *val2, int *num_elem)
{
	int i = threadIdx.x;
	val1[i] += val2[i]+1;
}

int main()
{
	return 0;
}

