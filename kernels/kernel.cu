#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>  
#include <stdlib.h>
#include <string.h>

#define Columnas 10
#define Filas 10
hipError_t addWithCuda(int* c, const int* a, unsigned int size);


__device__ unsigned int computeOutputEdge(int mask[][3], int vecinos[][3], int rows, int cols) {

	float result = 1;
	int sum = 0;

	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			float mul = mask[i][j] * vecinos[i][j];
			sum = sum + mul;
		}
	}
	result = abs(sum);
	return (int)result;
}

__global__ void bordes(int* val2, int* val1, int m, int n)
{

	int column = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	int myEdge[3][3] = { {0,1,0},{1,-4,1},{0,1,0} };
	//int filas = (sizeof(myMask)/sizeof(myMask[0]));

	if (row < m && column < n) {

		int thread_id1 = (row - 1) * n + (column - 1);
		int thread_id2 = (row - 1) * n + (column);
		int thread_id3 = (row - 1) * n + (column + 1);

		int thread_id4 = (row)* n + (column - 1);

		int thread_id5 = (row)* n + (column);

		int thread_id6 = (row)* n + (column + 1);

		int thread_id7 = (row + 1) * n + (column - 1);
		int thread_id8 = (row + 1) * n + (column);
		int thread_id9 = (row + 1) * n + (column + 1);

		//int my_val = val1[thread_id5];

		//printf("row: %d, \tcol: %d, \tvalor: %d\n", row, column, my_val);

		val2[thread_id5] = val1[thread_id5];

		if ((row > 0 && row < (m - 1)) && (column > 0 && column < (n - 1)))
		{
			int my_val0 = val1[thread_id1];
			int my_val2 = val1[thread_id2];
			int my_val3 = val1[thread_id3];
			int my_val4 = val1[thread_id4];
			int my_val5 = val1[thread_id5]; //doubly-subscripted access
			int my_val6 = val1[thread_id6];
			int my_val7 = val1[thread_id7];
			int my_val8 = val1[thread_id8];
			int my_val9 = val1[thread_id9];
			//printf("row: %d, col: %d, value: %d\n", row, column, my_val);

			int myMask2[3][3] = { {(my_val0),(my_val2),(my_val3)},
								 {(my_val4),(my_val5),(my_val6)},
								 {(my_val7),(my_val8),(my_val9)} };

			unsigned int output = computeOutputEdge(myEdge, myMask2, 3, 3);
			//printf("row: %d,\t col: %d,\t Valor Original: %d,\t Nuevo Valor: %d\n", row, column, my_val5,output);
			//printf("Salida: %d \n", output);
			//printf("Entro\n");
			val2[thread_id5] = output;
		}
	}
}

int main(int argc, char* argv[])
{
	//int a[Columnas * Filas] = { 0 };
	//int a[Columnas * Filas] = { 0,	0	,0	,97	,176,	176,	127,	0,0,	0,0	,0,	0	,108,	191,	191	,142,	0,	0,	0,0,	0,	0,	101,	191	,191,	136	,0,	0,	0, 99,	110,	110	,155,	191	,191,	169	,110,	110,	102,182	,191,	191	,191,	191,	191,	191	,191,	191,	189, 180,	191	,191,	191	,191,	191	,191,	191	,191,	187, 120,	134,	133	,165,	191	,191,	176,	133,	134,	124,0,	0,	0,	102,	191,	191	,136,	0,	0,	0,0	,0,	0,	107	,191,	191,	141,	0,	0,	0 ,0,	0	,0,	98	,177,	177,	129	,0	,0,	0 };
	//int c[Columnas * Filas] = { 0 };

	if(argc != 2) {
		printf("Usage: display_Image ImageToLoadandDisplay");
		return -1;
	}
		int m = 0;
		int n = 0;

		FILE* archivo = fopen(argv[1], "r");
		char* buffer = NULL;
		int* array = NULL;
		int j, c, x;

		array = (int*)realloc(NULL, sizeof(int));

		c = fgetc(archivo);
		buffer = (char*)realloc(NULL, sizeof(char));
		j = 0;
		x = 0;
		while (!feof(archivo))
		{
			if (c == '\t' || c == '\n')
			{
				array = (int*)realloc(array, (x + 1) * sizeof(int));
				array[x] = atoi(buffer);
				buffer = (char*)realloc(NULL, sizeof(char));
				j = 0;
				x++;
				if (c == '\n')
				{
					n++;
				}
			}
			else
			{
				buffer[j] = c;
				j++;
				buffer = (char*)realloc(buffer, (j + 1) * sizeof(char));
			}
			c = fgetc(archivo);
		}
		fclose(archivo);
		m = x / n;   
		
    int* prueba= (int*)realloc(NULL, (m*n)*sizeof(int));	


	FILE* ImagenO1 = fopen("ImagenOriginalAntes.txt", "w");
	int Col = 0;
	for (int j = 0; j < m *n; j++)
	{
		fprintf(ImagenO1, "%d\t", array[j]);
		if (n-1 == Col)
		{
			fprintf(ImagenO1, "\n");
			Col = -1;
		}
		Col++;
	}
	fclose(ImagenO1);


	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(prueba, array, Columnas * Filas);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed! Global");
		return 1;
	}

	/*Imprime Resultados
	FILE* R = fopen("R.txt", "w");
	
	for (int i = 0; i < Columnas * Filas; i++)
	{
		//printf("\nPosicion: %d\tValor Original: %d\tValor Procesado: %d",i,a[i],c[i]);	
		fprintf(R, "\nPosicion: %d\tValor Original: %d\tValor Procesado: %d", i, array[i], prueba[i]);
	}
	fclose(R);*/

	/*FILE* ImagenO = fopen("ImagenOriginal.txt", "w");
	Col = 0;
	for (int j = 0; j < Columnas * Filas; j++)
	{
		fprintf(ImagenO, "%d\t", a[j]);
		if (Columnas - 1 == Col)
		{
			fprintf(ImagenO, "\n");
			Col = -1;
		}
		Col++;
	}
	fclose(ImagenO);*/

	FILE* Imagen = fopen("ImagenProce.txt", "w");
	Col = 0;
	for (int a = 0; a < Columnas*Filas; a++)
	{
		fprintf(Imagen, "%d\t", prueba[a]);
		if (Columnas - 1 == Col)
		{
			fprintf(Imagen, "\n");
			Col = -1;
		}
		Col++;
	}
	fclose(Imagen);

	/*Imprime Resultados*/
	printf("Terminado");
	 // hipDeviceReset must be called before exiting in order for profiling and
	 // tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, unsigned int size)
{
	int* dev_a = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for two vectors (one input, one output)    .
	cudaStatus = hipMalloc((void**)& dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed! C");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! A");
		goto Error;
	}

	const dim3 gridSize = dim3(Columnas*Filas, Columnas*Filas);
	const dim3 gridThread = dim3(16, 16);
	// Launch a kernel on the GPU with one thread for each element.
	bordes << <gridSize, gridThread >> > (dev_c, dev_a, Columnas, Filas);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\naddKernel launch failed AQUI: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! CC");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);

	return cudaStatus;
}