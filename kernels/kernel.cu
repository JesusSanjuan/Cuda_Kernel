
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>  

typedef struct {

    int width;
    int height;
    int stride;
    float* elements;

    } Matrix;


__global__ void bordes( const int *A, const int *B, const  int filas, const int columnas, int LongVector )
{
	// Reserva de memoria
	int ** matriz = new int*[filas];
	for (int a = 0; a < filas; a++)
	{
		matriz[a] = new int[columnas];
	}
	// ingreso de valores
	int contador = 0;
	for (int a = 0; a < filas; a++)
	{
		for (int b = 0; b < columnas; b++)
		{
			contador = contador + 1;
			matriz[a][b] = A[contador];
		}
	}

	// Liberaci�n de la memoria
	/*for (int i = 0; i < filas; i++)
	{
		delete[] matriz[i];
	}

	delete[] matriz;*/


	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	int blockRow = blockIdx.y; 
	int blockCol = blockIdx.x;

}

/*__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
	int blockRow = blockIdx.y; int blockCol = blockIdx.x;
	Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
	float Cvalue = 0; // Variable para guardar resultado
	int row = threadIdx.y; int col = threadIdx.x;
	//Bucle para multiplicar submatrices Asubi y Bsubi
	for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

		Matrix Asub = GetSubMatrix(A, blockRow, m); // Obten Asub de A
		Matrix Bsub = GetSubMatrix(B, m, blockCol); // Obten Bsub de B
		// Declara y carga variables en memoria compartida
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
		As[row][col] = GetElement(Asub, row, col);
		Bs[row][col] = GetElement(Bsub, row, col);
		__syncthreads(); // Sincroniza para asegurar carga
		// Multiplica Asubi y Bsubi para actualizar Cvalue
		for (int e = 0; e < BLOCK_SIZE; ++e)
			Cvalue += As[row][e] * Bs[e][col];
		__syncthreads(); // Sincroniza para asegurar fin c�mputo previo }
		SetElement(Csub, row, col, Cvalue); // Escribe Csub a memoria global
	}*/

int main()
{
	return 0;
}

