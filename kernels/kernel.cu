#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>  
#include <stdlib.h>

hipError_t addWithCuda(int* c, const int* a, unsigned int size);


__device__ unsigned int computeOutputEdge(int mask[][3], int vecinos[][3], int rows, int cols) {

	float result = 1;
	int sum = 0;

	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			float mul = mask[i][j] * vecinos[i][j];
			sum = sum + mul;
		}
	}
	result = abs(sum);
	return (int)result;
}



__global__ void bordes(int* val2, int* val1, int m, int n)
{

	int column = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	int myEdge[3][3] = { {0,1,0},{1,-4,1},{0,1,0} };
	//int filas = (sizeof(myMask)/sizeof(myMask[0]));

	if (row < m && column < n) {

		int thread_id1 = (row - 1) * n + (column - 1);
		int thread_id2 = (row - 1) * n + (column);
		int thread_id3 = (row - 1) * n + (column + 1);

		int thread_id4 = (row)* n + (column - 1);

		int thread_id5 = (row)* n + (column);

		int thread_id6 = (row)* n + (column + 1);

		int thread_id7 = (row + 1) * n + (column - 1);
		int thread_id8 = (row + 1) * n + (column);
		int thread_id9 = (row + 1) * n + (column + 1);


		int my_val = val1[thread_id5];

		printf("row: %d, col: %d, value: %d\n", row, column, my_val);

		val2[thread_id5] = val1[thread_id5];

		if ((row > 0 && row < (m - 1)) && (column > 0 && column < (n - 1)))
		{
			int my_val0 = val1[thread_id1];
			int my_val2 = val1[thread_id2];
			int my_val3 = val1[thread_id3];
			int my_val4 = val1[thread_id4];
			int my_val5 = val1[thread_id5]; //doubly-subscripted access
			int my_val6 = val1[thread_id6];
			int my_val7 = val1[thread_id7];
			int my_val8 = val1[thread_id8];
			int my_val9 = val1[thread_id9];
			//printf("row: %d, col: %d, value: %d\n", row, column, my_val);

			int myMask2[3][3] = { {(my_val0),(my_val2),(my_val3)},
								 {(my_val4),(my_val5),(my_val6)},
								 {(my_val7),(my_val8),(my_val9)} };

			unsigned int output = computeOutputEdge(myEdge, myMask2, 3, 3);
			printf("output: %d", output);

			val2[thread_id5] = output;
		}
	}
}



/*__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}*/

int main()
{
	const int arraySize = 100;
	int a[arraySize] = { 0 };
	int c[arraySize] = { 0 };


	for (int i = 0; i < arraySize; i++)
	{
		int num = 1 + rand() % (256 - 1);
		a[i] = num;
	}
	printf("Valor: %d\n", a[0]);

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed! Global");
		return 1;
	}

	// printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",c[0], c[1], c[2], c[3], c[4]);

	 // hipDeviceReset must be called before exiting in order for profiling and
	 // tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, unsigned int size)
{
	int* dev_a = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for two vectors (one input, one output)    .
	cudaStatus = hipMalloc((void**)& dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed! C");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! A");
		goto Error;
	}


	// Launch a kernel on the GPU with one thread for each element.
	//addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
	bordes << <1, size >> > (dev_c, dev_a, 10, 10);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! CC");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);

	return cudaStatus;
}